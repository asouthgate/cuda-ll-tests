#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdlib>
#include <iostream>
#include <ctime>

#include "work.h"


__global__ void add(cxfloat *a, cxfloat *b, cxfloat *c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = work(a[i], b[i]);
}

void run_app(int N, int THREADS_PER_BLOCK) {


    cxfloat *a, *b, *c; // host copies of a, b, c
    cxfloat *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(cxfloat);
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Alloc space for host copies of a, b, c and setup input values
    a = (cxfloat *)malloc(size); init_arr(a, N);
    b = (cxfloat *)malloc(size); init_arr(b, N);
    c = (cxfloat *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    std::cerr << float(N)/THREADS_PER_BLOCK <<  std::endl;
    std::cerr << THREADS_PER_BLOCK << std::endl;
    int n_blocks = ceil( float(N)/THREADS_PER_BLOCK );
    add<<<n_blocks,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
//    hipError_t cudaerr = hipGetLastError();
//    if (cudaerr != hipSuccess)
//        printf("kernel launch failed with error \"%s\".\n",
//               hipGetErrorString(cudaerr));


    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    std::cout << a[55] << " " << b[55] << " " << c[55] << std::endl;

    
    cxfloat s = 0;
    for (int i = 0; i < N; ++i) {
//        std::cout << "ci " << a[i] << std::endl;
        s += c[i];
    }
    std::cout << "final answer " << s << std::endl;


    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);


}


int main(int argc, char* argv[]) {

    if (argc != 3) {

        fprintf(stderr,"%s: Error: 2 parameters expected. Found %d\n", argv[0], argc);

        return 1;

    }



    int N = atoi(argv[1]);
    int THREADS_PER_BLOCK = atoi(argv[2]);

    run_app(N, THREADS_PER_BLOCK);


    return 0;
}
